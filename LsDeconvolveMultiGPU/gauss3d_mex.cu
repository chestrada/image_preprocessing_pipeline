#include "hip/hip_runtime.h"
// gauss3d_mex_inplace.cu: In-place 3D Gaussian filter for MATLAB gpuArray input (no extra VRAM)
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <algorithm>

#define MAX_KERNEL_SIZE 51

#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) \
        mexErrMsgIdAndTxt("gauss3d:cuda", "CUDA error: %s", hipGetErrorString(err)); \
} while(0)

template <typename T>
__global__ void gauss1d_kernel(
    T* bufA, T* bufB, int nx, int ny, int nz,
    const float* kernel, int klen, int dim
) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz) return;

    int size[3] = {nx, ny, nz};
    int center = klen / 2;

    int idx = iz * nx * ny + iy * nx + ix;
    T val = 0;

    for (int k = 0; k < klen; ++k) {
        int offset = k - center;
        int ci = (dim == 0) ? ix + offset : (dim == 1) ? iy + offset : iz + offset;
        ci = min(max(ci, 0), size[dim] - 1);
        int cidx;
        if (dim == 0)
            cidx = iz * nx * ny + iy * nx + ci;
        else if (dim == 1)
            cidx = iz * nx * ny + ci * nx + ix;
        else
            cidx = ci * nx * ny + iy * nx + ix;
        val += bufA[cidx] * kernel[k];
    }
    bufB[idx] = val;
}

void make_gaussian_kernel(float sigma, float* kernel, int* klen) {
    int r = (int)ceilf(3.0f * sigma);
    *klen = 2*r + 1;
    if (*klen > MAX_KERNEL_SIZE)
        mexErrMsgIdAndTxt("gauss3d:kernel", "Kernel size exceeds MAX_KERNEL_SIZE (%d)", MAX_KERNEL_SIZE);
    float sum = 0.0f;
    for (int i = -r; i <= r; ++i) {
        kernel[i+r] = expf(-0.5f * (i*i) / (sigma*sigma));
        sum += kernel[i+r];
    }
    for (int i = 0; i < *klen; ++i)
        kernel[i] /= sum;
}

// Minimal VRAM: 1 buffer (input), 1 temp buffer.
template<typename T>
void run_gauss3d_inplace(T* bufA, int nx, int ny, int nz, float sigma[3]) {
    float *h_kernel = new float[MAX_KERNEL_SIZE];
    float *d_kernel;
    int klen;

    dim3 block(8,8,8);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y, (nz+block.z-1)/block.z);

    // Allocate one temp buffer for swap
    T* bufB;
    CUDA_CHECK(hipMalloc(&bufB, nx*ny*nz*sizeof(T)));

    T *src = bufA;
    T *dst = bufB;
    for (int dim = 0; dim < 3; ++dim) {
        make_gaussian_kernel(sigma[dim], h_kernel, &klen);
        CUDA_CHECK(hipMalloc(&d_kernel, klen * sizeof(float)));
        CUDA_CHECK(hipMemcpy(d_kernel, h_kernel, klen * sizeof(float), hipMemcpyHostToDevice));

        gauss1d_kernel<T><<<grid, block>>>(src, dst, nx, ny, nz, d_kernel, klen, dim);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipFree(d_kernel));

        // Swap: after last pass, dst will be bufA if odd, bufB if even
        T* tmp = src; src = dst; dst = tmp;
    }

    // If output is not in bufA, copy back from bufB (odd number of passes swaps back)
    if (src != bufA) {
        CUDA_CHECK(hipMemcpy(bufA, bufB, nx*ny*nz*sizeof(T), hipMemcpyDeviceToDevice));
    }
    CUDA_CHECK(hipFree(bufB));
    delete[] h_kernel;
}

// MEX entry (in-place modification of input gpuArray)
void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {
    mxInitGPU();

    if (nrhs < 2) mexErrMsgIdAndTxt("gauss3d:nrhs", "Need input array and sigma");

    // Use input gpuArray for in-place output
    mxGPUArray* img_gpu = mxGPUCreateFromMxArray(prhs[0]);
    const mwSize* sz = mxGPUGetDimensions(img_gpu);
    int nd = mxGPUGetNumberOfDimensions(img_gpu);
    if (nd != 3) mexErrMsgIdAndTxt("gauss3d:ndims", "Input must be 3D");
    int nx = (int)sz[0], ny = (int)sz[1], nz = (int)sz[2];

    float sigma[3];
    if (mxIsScalar(prhs[1])) sigma[0]=sigma[1]=sigma[2]=(float)mxGetScalar(prhs[1]);
    else if (mxGetNumberOfElements(prhs[1])==3) {
        double* ps = mxGetPr(prhs[1]);
        for(int i=0;i<3;i++) sigma[i]=(float)ps[i];
    } else mexErrMsgIdAndTxt("gauss3d:sigma", "Sigma must be scalar or length-3 vector");

    mxClassID cls = mxGPUGetClassID(img_gpu);
    void* ptr = mxGPUGetData(img_gpu);

    if (cls == mxSINGLE_CLASS)
        run_gauss3d_inplace<float>((float*)ptr, nx, ny, nz, sigma);
    else if (cls == mxDOUBLE_CLASS)
        run_gauss3d_inplace<double>((double*)ptr, nx, ny, nz, sigma);
    else mexErrMsgIdAndTxt("gauss3d:class", "Input must be single or double");

    // Return the (modified) input as output, **do not destroy img_gpu**
    plhs[0] = mxGPUCreateMxArrayOnGPU(img_gpu);
    // mxGPUDestroyGPUArray(img_gpu); // Do not destroy if returning as output
}
