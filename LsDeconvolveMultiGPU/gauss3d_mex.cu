#include "hip/hip_runtime.h"
// gauss3d_mex.cu - Optimized 3D Gaussian filtering (1 buffer, constant/shared memory kernel, in-place last axis)
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>
#include <cstdio>

#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) \
        mexErrMsgIdAndTxt("gauss3d:cuda", "CUDA error %s:%d: %s", __FILE__, __LINE__, hipGetErrorString(err)); \
} while(0)

#define MAX_KERNEL_SIZE 51  // practical upper bound for constant/shared memory

// ========================
// Constant memory for kernel
// ========================
__constant__ float const_kernel_f[MAX_KERNEL_SIZE];
__constant__ double const_kernel_d[MAX_KERNEL_SIZE];

// ========================
// Gaussian kernel creation
// ========================
template <typename T>
void make_gaussian_kernel(T sigma, int ksize, T* kernel) {
    int r = ksize / 2;
    double sum = 0.0;
    for (int i = -r; i <= r; ++i) {
        kernel[i + r] = (T)std::exp(-0.5 * (i * i) / (sigma * sigma));
        sum += kernel[i + r];
    }
    for (int i = 0; i < ksize; ++i) kernel[i] = (T)(kernel[i] / sum);
}

// =====================
// CUDA 1D convolution
// =====================
template <typename T, bool use_const_kernel>
__global__ void gauss1d_kernel(
    const T* src, T* dst,
    int nx, int ny, int nz,
    int klen, int axis)
{
    extern __shared__ T shared_kernel[];  // Kernel in shared memory
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nline, linelen;
    if (axis == 0) { linelen = nx; nline = ny * nz; }
    else if (axis == 1) { linelen = ny; nline = nx * nz; }
    else { linelen = nz; nline = nx * ny; }
    if (tid >= nline * linelen) return;

    // Shared memory: only one block copies kernel per launch
    if (!use_const_kernel && threadIdx.x < klen) {
        shared_kernel[threadIdx.x] = 0; // Just to suppress warnings for non-specializations
    }
    __syncthreads();

    int line = tid / linelen;
    int pos = tid % linelen;

    // Compute x/y/z for this line/position
    int x, y, z;
    if (axis == 0) {
        y = line % ny;
        z = line / ny;
        x = pos;
    } else if (axis == 1) {
        x = line % nx;
        z = line / nx;
        y = pos;
    } else {
        x = line % nx;
        y = line / nx;
        z = pos;
    }

    int idx = x + y * nx + z * nx * ny;
    int r = klen / 2;
    T acc = T(0);
    for (int s = 0; s < klen; ++s) {
        int offset = s - r;
        int xi = x, yi = y, zi = z;
        if (axis == 0) xi = min(max(x + offset, 0), nx - 1);
        if (axis == 1) yi = min(max(y + offset, 0), ny - 1);
        if (axis == 2) zi = min(max(z + offset, 0), nz - 1);
        int src_idx = xi + yi * nx + zi * nx * ny;
        T k;
        if constexpr (use_const_kernel) {
            if constexpr (std::is_same<T, float>::value)
                k = const_kernel_f[s];
            else
                k = const_kernel_d[s];
        } else {
            k = shared_kernel[s];
        }
        acc += src[src_idx] * k;
    }
    dst[idx] = acc;
}

// ===============================
// Host orchestration with 1 buffer
// ===============================
template <typename T>
void gauss3d_separable(
    T* input,           // in-place array (device ptr)
    T* buffer,          // extra buffer (device ptr, same size)
    int nx, int ny, int nz,
    const T sigma[3], const int ksize[3])
{
    size_t N = (size_t)nx * ny * nz;
    int max_klen = std::max({ksize[0], ksize[1], ksize[2]});
    if (max_klen > MAX_KERNEL_SIZE) {
        mexErrMsgIdAndTxt("gauss3d:ksize", "Kernel size exceeds MAX_KERNEL_SIZE (%d)", MAX_KERNEL_SIZE);
    }
    T* h_kernel = new T[max_klen];

    // Swap logic: input <-> buffer, last axis done in-place in input
    T* src = input;
    T* dst = buffer;

    for (int axis = 0; axis < 3; ++axis) {
        make_gaussian_kernel(sigma[axis], ksize[axis], h_kernel);

        int linelen = (axis == 0) ? nx : (axis == 1) ? ny : nz;
        int nline   = (axis == 0) ? ny * nz : (axis == 1) ? nx * nz : nx * ny;
        int total = linelen * nline;
        int block = 256;
        int grid = (total + block - 1) / block;

        // Last axis: do in-place in 'input' to save memory copy
        if (axis == 2) {
            dst = input;
        }

        // Use constant memory for kernel if possible
        if constexpr (std::is_same<T, float>::value) {
            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_kernel_f), h_kernel, ksize[axis] * sizeof(float), 0, hipMemcpyHostToDevice));
            gauss1d_kernel<T, true><<<grid, block, 0>>>(src, dst, nx, ny, nz, ksize[axis], axis);
        } else {
            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_kernel_d), h_kernel, ksize[axis] * sizeof(double), 0, hipMemcpyHostToDevice));
            gauss1d_kernel<T, true><<<grid, block, 0>>>(src, dst, nx, ny, nz, ksize[axis], axis);
        }
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        // Swap for next axis if not the last axis
        if (axis < 2) std::swap(src, dst);
    }
    delete[] h_kernel;
}

// ================
// MEX entry point
// ================
extern "C" void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {
    mxInitGPU();

    if (nrhs < 2) mexErrMsgIdAndTxt("gauss3d:nrhs", "Usage: gauss3d_mex(x, sigma [, kernel_size])");
    const mxGPUArray* img_gpu_const = mxGPUCreateFromMxArray(prhs[0]);
    mxGPUArray* img_gpu = const_cast<mxGPUArray*>(img_gpu_const); // No copy

    const mwSize* sz = mxGPUGetDimensions(img_gpu);
    int nd = mxGPUGetNumberOfDimensions(img_gpu);
    if (nd != 3) mexErrMsgIdAndTxt("gauss3d:ndims", "Input must be 3D.");
    int nx = (int)sz[0], ny = (int)sz[1], nz = (int)sz[2];

    double sigma_double[3];
    if (mxIsScalar(prhs[1])) {
        double v = mxGetScalar(prhs[1]);
        sigma_double[0] = sigma_double[1] = sigma_double[2] = v;
    } else if (mxGetNumberOfElements(prhs[1]) == 3) {
        double* s = mxGetPr(prhs[1]);
        for (int i = 0; i < 3; ++i) sigma_double[i] = s[i];
    } else {
        mexErrMsgIdAndTxt("gauss3d:sigma", "sigma must be scalar or 3-vector");
    }

    int ksize[3];
    if (nrhs >= 3 && !mxIsLogicalScalar(prhs[2])) {
        if (mxIsScalar(prhs[2])) {
            int k = (int)mxGetScalar(prhs[2]);
            ksize[0] = ksize[1] = ksize[2] = k;
        } else if (mxGetNumberOfElements(prhs[2]) == 3) {
            double* ks = mxGetPr(prhs[2]);
            for (int i = 0; i < 3; ++i) ksize[i] = (int)ks[i];
        } else {
            mexErrMsgIdAndTxt("gauss3d:kernel", "kernel_size must be scalar or 3-vector");
        }
    } else {
        for (int i = 0; i < 3; ++i)
            ksize[i] = 2 * (int)ceil(3.0 * sigma_double[i]) + 1;
    }

    mxClassID cls = mxGPUGetClassID(img_gpu);
    void* ptr = mxGPUGetData(img_gpu);

    // Allocate one extra buffer
    void* buffer = nullptr;
    size_t N = (size_t)nx * ny * nz;
    if (cls == mxSINGLE_CLASS) {
        CUDA_CHECK(hipMalloc(&buffer, N * sizeof(float)));
        float sigma[3]; for (int i = 0; i < 3; ++i) sigma[i] = (float)sigma_double[i];
        gauss3d_separable<float>((float*)ptr, (float*)buffer, nx, ny, nz, sigma, ksize);
    } else if (cls == mxDOUBLE_CLASS) {
        CUDA_CHECK(hipMalloc(&buffer, N * sizeof(double)));
        double sigma[3]; for (int i = 0; i < 3; ++i) sigma[i] = sigma_double[i];
        gauss3d_separable<double>((double*)ptr, (double*)buffer, nx, ny, nz, sigma, ksize);
    } else {
        mexErrMsgIdAndTxt("gauss3d:class", "Input must be single or double gpuArray");
    }
    CUDA_CHECK(hipFree(buffer));

    plhs[0] = mxGPUCreateMxArrayOnGPU(img_gpu);
    // Do not destroy img_gpu before return
}
