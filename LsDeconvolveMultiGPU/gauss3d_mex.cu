#include "hip/hip_runtime.h"
// gauss3d_mex.cu: In-place 3D Gaussian filter for MATLAB, with warnings and kernel normalization

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <algorithm>
#include <cstdlib>

#define MAX_KERNEL_SIZE 151

#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) \
        mexErrMsgIdAndTxt("gauss3d:cuda", "CUDA error: %s", hipGetErrorString(err)); \
} while(0)

// --------- KERNEL GENERATION & NORMALIZATION (float/double) ---------
template<typename T>
void make_gaussian_kernel(T sigma, int ksize, T* kernel) {
    int r = ksize / 2;
    T sum = 0;
    for (int i = -r; i <= r; ++i) {
        kernel[i + r] = exp(-0.5 * (i*i) / (sigma*sigma));
        sum += kernel[i + r];
    }
    // Normalize kernel so sum(kernel)==1
    for (int i = 0; i < ksize; ++i)
        kernel[i] /= sum;
}

// --------- CUDA KERNEL: Each block processes a line ---------
void warn_kernel_size(const double* sigma, const int* ksize, int do_warn) {
    for (int i = 0; i < 3; ++i) {
        int min_ksize = 2 * static_cast<int>(ceil(3.0 * sigma[i])) + 1;
        if (ksize[i] < min_ksize && do_warn) {
            mexWarnMsgIdAndTxt("gauss3d:kernelSizeTooSmall",
                "Kernel size for axis %d (%d) is too small for sigma=%.3f (recommended at least %d). Results may be inaccurate.\n"
                "To disable this warning, call gauss3d_mex(..., ..., ..., true) to disable warnings.",
                i+1, ksize[i], sigma[i], min_ksize);
        }
    }
}

template<typename T>
__global__ void gauss1d_lines_kernel(
    const T* __restrict__ data_in,
    T* data_out,
    const T* __restrict__ kernel, int klen,
    int nx, int ny, int nz,
    int dim, int line_len, int n_lines)
{
    int line_idx = blockIdx.x;
    int center = klen / 2;

    // Compute 3D coordinates of the start of this line
    int ix0 = 0, iy0 = 0, iz0 = 0;
    if (dim == 0) { iy0 = line_idx % ny; iz0 = line_idx / ny; }
    else if (dim == 1) { ix0 = line_idx % nx; iz0 = line_idx / nx; }
    else { ix0 = line_idx % nx; iy0 = line_idx / nx; }

    for (int idx_in_line = threadIdx.x; idx_in_line < line_len; idx_in_line += blockDim.x) {
        int ix = ix0, iy = iy0, iz = iz0;
        if (dim == 0) ix = idx_in_line;
        else if (dim == 1) iy = idx_in_line;
        else iz = idx_in_line;

        double val = 0.0;
        for (int k = 0; k < klen; ++k) {
            int offset = k - center;
            int ci = idx_in_line + offset;
            ci = min(max(ci, 0), line_len - 1);

            int cx = ix0, cy = iy0, cz = iz0;
            if (dim == 0) cx = ci;
            else if (dim == 1) cy = ci;
            else cz = ci;
            int in_idx = cz * nx * ny + cy * nx + cx;

            val += static_cast<double>(data_in[in_idx]) * static_cast<double>(kernel[k]);
        }

        int out_idx = iz * nx * ny + iy * nx + ix;
        data_out[out_idx] = static_cast<T>(val);
    }
}

// --------- SEPARABLE GAUSS 3D (in-place, parallelized) ---------
template<typename T>
void run_gauss3d_inplace(T* buf, int nx, int ny, int nz, const T sigma[3], const int ksize[3]) {
    size_t nvox = nx * ny * nz;
    T* h_kernel = new T[MAX_KERNEL_SIZE];
    T* d_kernel;

    // Allocate a temp buffer on the GPU for out-of-place computation
    T* buf_tmp;
    CUDA_CHECK(hipMalloc(&buf_tmp, nvox * sizeof(T)));

    T* src = buf;
    T* dst = buf_tmp;

    for (int dim = 0; dim < 3; ++dim) {
        int klen = ksize[dim];
        if (klen > MAX_KERNEL_SIZE)
            mexErrMsgIdAndTxt("gauss3d:kernel", "Kernel size exceeds MAX_KERNEL_SIZE (%d)", MAX_KERNEL_SIZE);
        make_gaussian_kernel(sigma[dim], klen, h_kernel);
        CUDA_CHECK(hipMalloc(&d_kernel, klen * sizeof(T)));
        CUDA_CHECK(hipMemcpy(d_kernel, h_kernel, klen * sizeof(T), hipMemcpyHostToDevice));

        int n_lines, line_len;
        if (dim == 0) { n_lines = ny * nz; line_len = nx; }
        else if (dim == 1) { n_lines = nx * nz; line_len = ny; }
        else { n_lines = nx * ny; line_len = nz; }

        int block_size = (line_len < 256) ? line_len : 256;
        dim3 block(block_size);
        dim3 grid(n_lines);

        gauss1d_lines_kernel<T><<<grid, block>>>(src, dst, d_kernel, klen, nx, ny, nz, dim, line_len, n_lines);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipFree(d_kernel));

        // Swap src and dst for the next pass
        T* tmp = src;
        src = dst;
        dst = tmp;
    }

    // After 3 passes (odd), src points to buf_tmp, copy back if needed
    if (src != buf) {
        CUDA_CHECK(hipMemcpy(buf, buf_tmp, nvox * sizeof(T), hipMemcpyDeviceToDevice));
    }

    CUDA_CHECK(hipFree(buf_tmp));
    delete[] h_kernel;
}

// --------- MEX ENTRY ---------
extern "C"
void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {
    mxInitGPU();

    if (nrhs < 2) mexErrMsgIdAndTxt("gauss3d:nrhs", "Usage: gauss3d_mex(x, sigma [, kernel_size, disable_warning])");

    // Parse input array (in-place)
    const mxGPUArray* img_gpu_const = mxGPUCreateFromMxArray(prhs[0]);
    mxGPUArray* img_gpu = const_cast<mxGPUArray*>(img_gpu_const);
    const mwSize* sz = mxGPUGetDimensions(img_gpu);
    int nd = mxGPUGetNumberOfDimensions(img_gpu);
    if (nd != 3) mexErrMsgIdAndTxt("gauss3d:ndims", "Input must be 3D");
    int nx = (int)sz[0], ny = (int)sz[1], nz = (int)sz[2];

    // --- Parse sigma ---
    double sigma_double[3];
    if (mxIsScalar(prhs[1])) {
        double v = mxGetScalar(prhs[1]);
        sigma_double[0] = sigma_double[1] = sigma_double[2] = v;
    } else if (mxGetNumberOfElements(prhs[1]) == 3) {
        double* s = mxGetPr(prhs[1]);
        for(int i=0; i<3; ++i) sigma_double[i] = s[i];
    } else {
        mexErrMsgIdAndTxt("gauss3d:sigma", "sigma must be scalar or 3-vector");
    }

    // --- Parse kernel_size (optional) ---
    int ksize[3];
    if (nrhs >= 3 && !mxIsLogicalScalar(prhs[2])) {
        if (mxIsScalar(prhs[2])) {
            int k = (int)mxGetScalar(prhs[2]);
            ksize[0] = ksize[1] = ksize[2] = k;
        } else if (mxGetNumberOfElements(prhs[2]) == 3) {
            double* ks = mxGetPr(prhs[2]);
            for(int i=0; i<3; ++i) ksize[i] = (int)ks[i];
        } else {
            mexErrMsgIdAndTxt("gauss3d:kernel", "kernel_size must be scalar or 3-vector");
        }
    } else {
        for(int i=0; i<3; ++i)
            ksize[i] = 2 * (int)ceil(3.0 * sigma_double[i]) + 1;
    }

    // --- Check warning preference (4th argument, optional) ---
    int do_warn = 1;
    if (nrhs >= 4 && mxIsLogicalScalar(prhs[3])) {
        do_warn = !mxIsLogicalScalarTrue(prhs[3]);
    } else {
        // Check environment variable (for backward compatibility)
        const char* warn_env = std::getenv("GAUSS3D_WARN_KSIZE");
        if (warn_env && warn_env[0] == '0') do_warn = 0;
    }
    warn_kernel_size(sigma_double, ksize, do_warn);

    // --- Convert sigma to float or double as appropriate ---
    mxClassID cls = mxGPUGetClassID(img_gpu);
    void* ptr = mxGPUGetData(img_gpu);

    if (cls == mxSINGLE_CLASS) {
        float sigma[3];
        for (int i = 0; i < 3; ++i) sigma[i] = (float)sigma_double[i];
        run_gauss3d_inplace<float>((float*)ptr, nx, ny, nz, sigma, ksize);
    } else if (cls == mxDOUBLE_CLASS) {
        double sigma[3];
        for (int i = 0; i < 3; ++i) sigma[i] = sigma_double[i];
        run_gauss3d_inplace<double>((double*)ptr, nx, ny, nz, sigma, ksize);
    } else {
        mexErrMsgIdAndTxt("gauss3d:class", "Input must be single or double");
    }

    // Return the modified input as output
    plhs[0] = mxGPUCreateMxArrayOnGPU(img_gpu);
    // Do not destroy img_gpu before return
}
