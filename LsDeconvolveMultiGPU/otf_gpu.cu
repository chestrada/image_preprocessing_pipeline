#include "hip/hip_runtime.h"
/*==============================================================================
  otf_gpu_mex.cu
  ------------------------------------------------------------------------------
  Compute 3-D Optical Transfer Function (OTF) on the GPU, using a user-provided
  complex gpuArray as buffer and output.

  Usage in MATLAB (all gpuArray, single):
      otf = otf_gpu_mex(psf, [nx ny nz], ..., buffer);

  Inputs
  ──────
    psf        : 3-D unshifted PSF (Y×X×Z)       single gpuArray, real
    fft_shape  : [nx ny nz]                      double, output size
    buffer     : 3-D single complex gpuArray     used as internal + output buffer

  Output
  ──────
    otf        : 3-D complex single gpuArray     (buffer, filled with OTF)
==============================================================================*/

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// ──────────────── Error-handling helpers ────────────────
#define CUDA_CHECK(err) \
    if ((err) != hipSuccess) \
        mexErrMsgIdAndTxt("otf_gpu_mex:CUDA", "CUDA error %s:%d: %s", \
                          __FILE__, __LINE__, hipGetErrorString(err));

#define CUFFT_CHECK(err) \
    if ((err) != HIPFFT_SUCCESS) \
        mexErrMsgIdAndTxt("otf_gpu_mex:CUFFT", "cuFFT error %s:%d: %d", \
                          __FILE__, __LINE__, int(err));

// ──────────────── Kernel: 0-filled, centred pad + axis swap ────────────────
__global__ void pad_center_swap(
    const float *src, size_t sx, size_t sy, size_t sz,
    float2 *dst,       size_t dx, size_t dy, size_t dz,
    ptrdiff_t pre_x, ptrdiff_t pre_y, ptrdiff_t pre_z)
{
    size_t z = blockIdx.x * blockDim.x + threadIdx.x;  // NOTE: Z fastest for cuFFT
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t x = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= dx || y >= dy || z >= dz) return;

    ptrdiff_t sx_i = ptrdiff_t(x) - pre_x;
    ptrdiff_t sy_i = ptrdiff_t(y) - pre_y;
    ptrdiff_t sz_i = ptrdiff_t(z) - pre_z;

    size_t dst_idx = x + dx * (y + dy * z); // C-order

    if (sx_i >= 0 && sx_i < ptrdiff_t(sx) &&
        sy_i >= 0 && sy_i < ptrdiff_t(sy) &&
        sz_i >= 0 && sz_i < ptrdiff_t(sz))
    {
        size_t src_idx = size_t(sx_i) + sx * (size_t(sy_i) + sy * size_t(sz_i));
        dst[dst_idx].x = src[src_idx];
        dst[dst_idx].y = 0.f;
    }
    else
    {
        dst[dst_idx].x = 0.f;
        dst[dst_idx].y = 0.f;
    }
}

// ──────────────── Kernel: full 3-D ifftshift ────────────────
__device__ __forceinline__ int ifftshift_i(int i, int dim)
{
    int s = dim / 2;
    int j = i + s;
    return (j >= dim) ? j - dim : j;
}

__global__ void ifftshift3D(float2 *v, int nx, int ny, int nz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int i2 = ifftshift_i(i, nx);
    int j2 = ifftshift_i(j, ny);
    int k2 = ifftshift_i(k, nz);

    // swap once per pair
    if ( (i  < i2) ||
         (i == i2 && j  < j2) ||
         (i == i2 && j == j2 && k < k2) )
    {
        size_t a = i  + nx * (j  + ny * k );
        size_t b = i2 + nx * (j2 + ny * k2);
        float2 tmp = v[a];
        v[a] = v[b];
        v[b] = tmp;
    }
}

// ───────────────────────── MEX entry ─────────────────────────
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, const mxArray *prhs[])
{
    // 4th argument required: user-provided buffer
    if (nrhs != 3 && nrhs != 4)
        mexErrMsgIdAndTxt("otf_gpu_mex:nrhs", "Three or four inputs required (psf, fft_shape, [unused], buffer).");
    if (nlhs != 1)
        mexErrMsgIdAndTxt("otf_gpu_mex:nlhs", "One output (otf) required.");

    mxInitGPU();

    // ---- PSF ----
    const mxGPUArray *psf = mxGPUCreateFromMxArray(prhs[0]);
    if (mxGPUGetClassID(psf) != mxSINGLE_CLASS || mxGPUGetNumberOfDimensions(psf) != 3)
        mexErrMsgIdAndTxt("otf_gpu_mex:psf", "psf must be 3-D single gpuArray.");

    const mwSize *pd = mxGPUGetDimensions(psf);
    size_t sx = pd[0], sy = pd[1], sz = pd[2];
    const float *d_psf = static_cast<const float*>(mxGPUGetDataReadOnly(psf));

    // ---- fft_shape ----
    if (!mxIsDouble(prhs[1]) || mxGetNumberOfElements(prhs[1]) != 3)
        mexErrMsgIdAndTxt("otf_gpu_mex:fftshape", "fft_shape must be [nx ny nz] double.");

    double *sh = mxGetPr(prhs[1]);
    size_t dx = size_t(sh[0]), dy = size_t(sh[1]), dz = size_t(sh[2]);
    if (!dx || !dy || !dz)
        mexErrMsgIdAndTxt("otf_gpu_mex:fftshape", "fft_shape must be positive.");

    mwSize odims[3] = { mwSize(dx), mwSize(dy), mwSize(dz) };

    // ---- User-provided buffer ----
    if (nrhs < 4)
        mexErrMsgIdAndTxt("otf_gpu_mex:buffer", "User-provided buffer (complex single gpuArray) required as 4th argument.");

    mxGPUArray *user_buffer = const_cast<mxGPUArray*>(mxGPUCreateFromMxArray(prhs[3]));
    if (mxGPUGetClassID(user_buffer) != mxSINGLE_CLASS ||
        !mxGPUGetIsComplex(user_buffer) ||
        mxGPUGetNumberOfDimensions(user_buffer) != 3)
    {
        mxGPUDestroyGPUArray(user_buffer);
        mxGPUDestroyGPUArray(psf);
        mexErrMsgIdAndTxt("otf_gpu_mex:buffer", "Buffer must be 3-D complex single gpuArray.");
    }

    const mwSize *bdims = mxGPUGetDimensions(user_buffer);
    if (bdims[0] != dx || bdims[1] != dy || bdims[2] != dz)
    {
        mxGPUDestroyGPUArray(user_buffer);
        mxGPUDestroyGPUArray(psf);
        mexErrMsgIdAndTxt("otf_gpu_mex:buffer", "Buffer must match fft_shape.");
    }

    float2 *d_otf = static_cast<float2*>(mxGPUGetData(user_buffer));

    // ---- Zero-pad & centre PSF into buffer ----
    dim3 blk(8,8,8);
    dim3 grd( (dz+blk.x-1)/blk.x,
              (dy+blk.y-1)/blk.y,
              (dx+blk.z-1)/blk.z );
    ptrdiff_t pre_x = (ptrdiff_t)( (dx - sx) / 2 );
    ptrdiff_t pre_y = (ptrdiff_t)( (dy - sy) / 2 );
    ptrdiff_t pre_z = (ptrdiff_t)( (dz - sz) / 2 );

    pad_center_swap<<<grd, blk>>>(d_psf, sx, sy, sz,
                                  d_otf, dx, dy, dz,
                                  pre_x, pre_y, pre_z);
    CUDA_CHECK(hipGetLastError());

    // ---- ifftshift in place (buffer) ----
    dim3 grd2( (dx+blk.x-1)/blk.x,
               (dy+blk.y-1)/blk.y,
               (dz+blk.z-1)/blk.z );
    ifftshift3D<<<grd2, blk>>>(d_otf, (int)dx, (int)dy, (int)dz);
    CUDA_CHECK(hipGetLastError());

    // ---- 3-D FFT in place (buffer) ----
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan3d((int)dz, (int)dy, (int)dx, HIPFFT_C2C));
    CUFFT_CHECK(hipfftExecC2C(plan,
                 reinterpret_cast<hipfftComplex*>(d_otf),
                 reinterpret_cast<hipfftComplex*>(d_otf),
                 HIPFFT_FORWARD));
    CUFFT_CHECK(hipfftDestroy(plan));

    CUDA_CHECK(hipDeviceSynchronize());

    // ---- Return the user-provided buffer as the OTF (MATLAB gpuArray) ----
    plhs[0] = mxGPUCreateMxArrayOnGPU(user_buffer);

    // ---- Free input and local references (but not the user buffer, which is output) ----
    mxGPUDestroyGPUArray(psf);
    mxGPUDestroyGPUArray(user_buffer); // safe to destroy here, as output is a new mxArray referencing the same GPU data
}
