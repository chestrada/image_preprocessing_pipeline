#include "hip/hip_runtime.h"
/*
    gauss3d_gpu.cu

    High-performance 3D Gaussian filtering for MATLAB gpuArray inputs.

    ----------------------------------------------------------------------------
    Author:       Keivan Moradi (with assistance from ChatGPT v4.1, 2025)
    License:      GPL v3
    ----------------------------------------------------------------------------

    Overview:
    ---------
    This MEX function implements a fast, memory-efficient, block-wise **3D Gaussian filter**
    for single-precision (`single`) 3D gpuArray data in MATLAB, using CUDA for GPU acceleration.
    It is **API-compatible** with MATLAB's `imgaussfilt3`, but is highly optimized
    for batch processing and large volumes, and designed to be integrated into GPU deconvolution pipelines.

    Key Features:
    -------------
      - **Heavy CUDA optimization**: Performs separable convolution along all 3 axes using constant-memory kernels, and launches tuned CUDA kernels for maximum performance.
      - **Workspace control**: Accepts user-provided block padding and kernel size to allow batch-wise processing (important for large volumes or integration in multi-step GPU workflows).
      - **OOM-resilient**: Attempts memory allocation with automatic retries and helpful warnings when out-of-memory occurs.
      - **MATLAB gpuArray interface**: Input and output are both MATLAB `gpuArray(single)` objects, fully compatible with native MATLAB workflows.
      - **Flexible sigma and kernel size**: Accepts scalar or vector `sigma` and kernel size for anisotropic filtering.
      - **Open source, GPL v3**.

    Differences from MATLAB's `imgaussfilt3`:
    -----------------------------------------
      1. **Much faster** on large data: Algorithm is hand-optimized for GPU with memory reuse and minimal transfers.
      2. **External workspace control**: Padding/batching is managed outside the function, making it suitable for tiled processing during deconvolution or large-scale pipelines.
      3. **Separable convolution**: Uses 1D convolutions in 3 passes, exploiting constant memory for kernel coefficients.
      4. **Direct gpuArray support**: Does not require conversion or intermediate CPU copies.

    Usage Example (in MATLAB):
    --------------------------
        x = gpuArray(single(randn(128,128,64)));
        y = gauss3d_gpu(x, 2.0);               % Isotropic sigma
        y = gauss3d_gpu(x, [2 1 4], [9 5 15]); % Anisotropic sigma & kernel size

    Notes:
    ------
      - Input must be a 3D `gpuArray` of single precision.
      - Designed for block-wise and pipelined use, e.g., in deconvolution, denoising, or pre-processing.
      - All main computation is performed on the GPU with minimal synchronization overhead.

    Acknowledgments:
    ----------------
      - Original algorithm and MEX/CUDA optimizations by Keivan Moradi.
      - ChatGPT (OpenAI GPT-4.1, 2025) provided structural and code review assistance.

*/

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <thread>   // For std::this_thread::sleep_for
#include <chrono>   // For std::chrono::milliseconds

#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA error %s:%d: %s", __FILE__, __LINE__, hipGetErrorString(err)); \
    } \
} while(0)

#define MAX_KERNEL_SIZE 51
__constant__ float const_kernel_f[MAX_KERNEL_SIZE];

// Gaussian kernel creation (host)
void make_gaussian_kernel(float sigma, int ksize, float* kernel) {
    int r = ksize / 2;
    double sum = 0.0;
    for (int i = -r; i <= r; ++i) {
        kernel[i + r] = static_cast<float>(std::exp(-0.5 * (i * i) / (sigma * sigma)));
        sum += kernel[i + r];
    }
    for (int i = 0; i < ksize; ++i)
        kernel[i] = static_cast<float>(kernel[i] / sum);
}

// CUDA 1D convolution kernel for float (with restrict)
__global__ void gauss1d_kernel_const_float(
    const float* __restrict__ src, float* __restrict__ dst,
    size_t nx, size_t ny, size_t nz,
    int klen, int axis)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nline, linelen;
    if (axis == 0) { linelen = nx; nline = ny * nz; }
    else if (axis == 1) { linelen = ny; nline = nx * nz; }
    else { linelen = nz; nline = nx * ny; }
    if (tid >= nline * linelen) return;

    size_t line = tid / linelen;
    size_t pos = tid % linelen;

    size_t x, y, z;
    if (axis == 0) {
        y = line % ny;
        z = line / ny;
        x = pos;
    } else if (axis == 1) {
        x = line % nx;
        z = line / nx;
        y = pos;
    } else {
        x = line % nx;
        y = line / nx;
        z = pos;
    }

    size_t idx = x + y * nx + z * nx * ny;
    int r = klen / 2;
    float acc = 0.0f;
    for (int s = 0; s < klen; ++s) {
        int offset = s - r;
        int xi = static_cast<int>(x);
        int yi = static_cast<int>(y);
        int zi = static_cast<int>(z);
        if (axis == 0) xi = min(max(static_cast<int>(x) + offset, 0), static_cast<int>(nx) - 1);
        if (axis == 1) yi = min(max(static_cast<int>(y) + offset, 0), static_cast<int>(ny) - 1);
        if (axis == 2) zi = min(max(static_cast<int>(z) + offset, 0), static_cast<int>(nz) - 1);
        size_t src_idx = xi + yi * nx + zi * nx * ny;
        acc += src[src_idx] * const_kernel_f[s];
    }
    dst[idx] = acc;
}

// Host orchestration for float
void gauss3d_separable_float(
    float* input,
    float* buffer,
    size_t nx, size_t ny, size_t nz,
    const float sigma[3], const int ksize[3],
    bool* error_flag)
{
    int max_klen = std::max({ksize[0], ksize[1], ksize[2]});
    if (max_klen > MAX_KERNEL_SIZE) {
        mexWarnMsgIdAndTxt("gauss3d_gpu:ksize", "Kernel size exceeds MAX_KERNEL_SIZE (%d)", MAX_KERNEL_SIZE);
        if (error_flag) *error_flag = true;
        return;
    }
    float* h_kernel = new float[max_klen];
    float* src = input;
    float* dst = buffer;
    bool local_error = false;

    // --- Use hipOccupancyMaxPotentialBlockSize for kernel launch tuning ---
    int minGrid, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGrid, &blockSize, gauss1d_kernel_const_float, 0, 0);

    for (int axis = 0; axis < 3; ++axis) {
        make_gaussian_kernel(sigma[axis], ksize[axis], h_kernel);
        hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(const_kernel_f), h_kernel, ksize[axis] * sizeof(float), 0, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA memcpyToSymbol error: %s", hipGetErrorString(err));
            local_error = true;
            break;
        }
        size_t linelen = (axis == 0) ? nx : (axis == 1) ? ny : nz;
        size_t nline   = (axis == 0) ? ny * nz : (axis == 1) ? nx * nz : nx * ny;
        size_t total = linelen * nline;
        int grid = static_cast<int>((total + blockSize - 1) / blockSize);

        gauss1d_kernel_const_float<<<grid, blockSize, 0>>>(
            src, dst, nx, ny, nz, ksize[axis], axis);

        err = hipGetLastError();
        if (err != hipSuccess) {
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA kernel launch error: %s", hipGetErrorString(err));
            local_error = true;
            break;
        }
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA device synchronize error: %s", hipGetErrorString(err));
            local_error = true;
            break;
        }
        std::swap(src, dst);
    }

    if (!local_error && src != input) {
        hipError_t err = hipMemcpy(input, src, nx * ny * nz * sizeof(float), hipMemcpyDeviceToDevice);
        if (err != hipSuccess) {
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA memcpy result error: %s", hipGetErrorString(err));
            local_error = true;
        }
    }

    delete[] h_kernel;
    if (error_flag) *error_flag = local_error;
}

// ================
// MEX entry point
// ================
extern "C" void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {
    mxInitGPU();
    float* buffer = nullptr;
    bool error_flag = false;
    mxGPUArray* img_gpu = nullptr;
    mxGPUArray* out_gpu = nullptr;

    try {
        if (nrhs < 2)
            mexErrMsgIdAndTxt("gauss3d_gpu:", "Usage: gauss3d_gpu(x, sigma [, kernel_size])");

        img_gpu = (mxGPUArray*)mxGPUCreateFromMxArray(prhs[0]);
        const mwSize* sz = mxGPUGetDimensions(img_gpu);
        int nd = mxGPUGetNumberOfDimensions(img_gpu);
        if (nd != 3)
            mexErrMsgIdAndTxt("gauss3d_gpu:", "Input must be 3D.");

        size_t nx = (size_t)sz[0], ny = (size_t)sz[1], nz = (size_t)sz[2];
        size_t N = nx * ny * nz;
        mxClassID cls = mxGPUGetClassID(img_gpu);
        void* ptr = mxGPUGetData(img_gpu);

        if (cls != mxSINGLE_CLASS)
            mexErrMsgIdAndTxt("gauss3d_gpu:", "Input must be single-precision gpuArray");

        double sigma_double[3];
        if (mxIsScalar(prhs[1])) {
            double v = mxGetScalar(prhs[1]);
            sigma_double[0] = sigma_double[1] = sigma_double[2] = v;
        } else if (mxGetNumberOfElements(prhs[1]) == 3) {
            double* s = mxGetPr(prhs[1]);
            for (int i = 0; i < 3; ++i) sigma_double[i] = s[i];
        } else {
            mexErrMsgIdAndTxt("gauss3d_gpu:", "sigma must be scalar or 3-vector");
        }

        int ksize[3];
        if (nrhs >= 3 && !mxIsLogicalScalar(prhs[2])) {
            if (mxIsEmpty(prhs[2])) {
                for (int i = 0; i < 3; ++i)
                    ksize[i] = 2 * (int)ceil(3.0 * sigma_double[i]) + 1;
            } else if (mxIsScalar(prhs[2])) {
                int k = (int)mxGetScalar(prhs[2]);
                ksize[0] = ksize[1] = ksize[2] = k;
            } else if (mxGetNumberOfElements(prhs[2]) == 3) {
                double* ks = mxGetPr(prhs[2]);
                for (int i = 0; i < 3; ++i) ksize[i] = (int)ks[i];
            } else {
                mexErrMsgIdAndTxt("gauss3d_gpu:", "kernel_size must be scalar or 3-vector");
            }
        } else {
            for (int i = 0; i < 3; ++i)
                ksize[i] = 2 * (int)ceil(3.0 * sigma_double[i]) + 1;
        }

        // --------- Now that all validation is done, allocate GPU buffer --------
        int max_retries = 2;
        int retries = 0;
        hipError_t alloc_err;
        while (retries < max_retries) {
            alloc_err = hipMalloc(&buffer, N * sizeof(float));
            if (alloc_err == hipSuccess && buffer != nullptr)
                break;
            size_t free_bytes = 0, total_bytes = 0;
            hipMemGetInfo(&free_bytes, &total_bytes);
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda",
                "gauss3d_gpu: CUDA OOM: Tried to allocate %.2f MB (Free: %.2f MB). Attempt %d/%d.",
                N * sizeof(float) / 1024.0 / 1024.0,
                free_bytes / 1024.0 / 1024.0,
                retries + 1, max_retries);
            hipDeviceSynchronize();
            std::this_thread::sleep_for(std::chrono::milliseconds(1000));
            retries++;
        }
        if (alloc_err != hipSuccess || !buffer) {
            mexErrMsgIdAndTxt("gauss3d_gpu:cuda",
                "gauss3d_gpu: CUDA OOM: Could not allocate workspace buffer (%.2f MB) after %d attempts.",
                N * sizeof(float) / 1024.0 / 1024.0, max_retries);
        }

        // --------- Allocate a fresh output gpuArray (in-place copy is unsafe) ---------
        out_gpu = mxGPUCreateGPUArray(3, sz, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
        float* dst_ptr = static_cast<float*>(mxGPUGetData(out_gpu));
        CUDA_CHECK(hipMemcpy(dst_ptr, ptr, N * sizeof(float), hipMemcpyDeviceToDevice));

        float sigma[3] = { (float)sigma_double[0], (float)sigma_double[1], (float)sigma_double[2] };
        gauss3d_separable_float(dst_ptr, buffer, nx, ny, nz, sigma, ksize, &error_flag);

        // Final sync before returning
        CUDA_CHECK(hipDeviceSynchronize());

        // Return as mxArray
        plhs[0] = mxGPUCreateMxArrayOnGPU(out_gpu);

    } catch (...) {
        mexPrintf("gauss3d_gpu: Unknown error! Possible OOM or kernel failure.\n");
        error_flag = true;
    }

    // ----------- CLEANUP (always reached) --------------
    if (buffer)
        hipFree(buffer);
    if (img_gpu)
        mxGPUDestroyGPUArray(img_gpu);
    if (out_gpu)
        mxGPUDestroyGPUArray(out_gpu);
}
