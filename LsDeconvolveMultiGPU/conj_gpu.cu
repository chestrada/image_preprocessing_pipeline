#include "hip/hip_runtime.h"
/*==============================================================================
  conj_gpu.cu
  ------------------------------------------------------------------------------
  Compute the complex conjugate of an existing single-precision OTF on the GPU.

  Usage in MATLAB (gpuArray):
      otf_conj = conj_gpu(otf);

  Input
  ──────
    otf         : 3-D complex single gpuArray (output of otf_gpu)

  Output
  ──────
    otf_conj    : 3-D complex single gpuArray  (same size as input)
==============================================================================*/

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>

// ─────────────── Error macro ───────────────
#define CUDA_CHECK(e) \
    if ((e) != hipSuccess) \
        mexErrMsgIdAndTxt("conj_gpu:CUDA", "CUDA error %s:%d: %s", \
                          __FILE__, __LINE__, hipGetErrorString(e));

// ─────────────── Kernel ───────────────
__global__ void conj_kernel(const float2 *src, float2 *dst, size_t N)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        dst[idx].x = src[idx].x;
        dst[idx].y = -src[idx].y;
    }
}

// ─────────────── MEX entry ───────────────
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, const mxArray *prhs[])
{
    if (nrhs != 1)
        mexErrMsgIdAndTxt("conj_gpu:nrhs", "One input (otf) required.");
    if (nlhs != 1)
        mexErrMsgIdAndTxt("conj_gpu:nlhs", "One output (otf_conj) required.");

    mxInitGPU();

    const mxGPUArray *otf = mxGPUCreateFromMxArray(prhs[0]);

    if (mxGPUGetClassID(otf) != mxSINGLE_CLASS)
        mexErrMsgIdAndTxt("conj_gpu:type", "Input must be single precision.");
    if (!mxGPUIsComplex(otf))
        mexErrMsgIdAndTxt("conj_gpu:complex", "Input must be complex.");

    // Compute element count manually
    size_t N = 1;
    const mwSize *dims = mxGPUGetDimensions(otf);
    mwSize nd = mxGPUGetNumberOfDimensions(otf);
    for (mwSize i = 0; i < nd; ++i) N *= dims[i];

    const float2 *d_in = static_cast<const float2*>(mxGPUGetDataReadOnly(otf));

    mxGPUArray *out = mxGPUCreateGPUArray(nd, dims,
                                          mxSINGLE_CLASS, mxCOMPLEX,
                                          MX_GPU_DO_NOT_INITIALIZE);
    float2 *d_out = static_cast<float2*>(mxGPUGetData(out));

    dim3 blk(256);
    dim3 grd((N + blk.x - 1) / blk.x);
    conj_kernel<<<grd, blk>>>(d_in, d_out, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    plhs[0] = mxGPUCreateMxArrayOnGPU(out);
    mxGPUDestroyGPUArray(otf);
}
