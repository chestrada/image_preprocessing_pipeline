#include "hip/hip_runtime.h"
// File: conv3d_mex.cu
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>

__global__ void conv3d_single(
    const float* img, const float* kernel, float* out,
    int nx, int ny, int nz, int kx, int ky, int kz)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x >= nx || y >= ny || z >= nz) return;

    int kx2 = kx / 2;
    int ky2 = ky / 2;
    int kz2 = kz / 2;

    float acc = 0.0f;
    for (int dz = 0; dz < kz; dz++) {
        int iz = z + dz - kz2;
        // Clamp for replicate boundary
        iz = iz < 0 ? 0 : (iz >= nz ? nz - 1 : iz);
        for (int dy = 0; dy < ky; dy++) {
            int iy = y + dy - ky2;
            iy = iy < 0 ? 0 : (iy >= ny ? ny - 1 : iy);
            for (int dx = 0; dx < kx; dx++) {
                int ix = x + dx - kx2;
                ix = ix < 0 ? 0 : (ix >= nx ? nx - 1 : ix);

                // MATLAB column-major order for 3D: (ix,iy,iz) = [x,y,z]
                int img_idx = ix + iy * nx + iz * nx * ny;
                int ker_idx = dx + dy * kx + dz * kx * ky; // C-style flat

                acc += img[img_idx] * kernel[ker_idx];
            }
        }
    }
    // Write output
    int out_idx = x + y * nx + z * nx * ny;
    out[out_idx] = acc;
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    mxInitGPU();

    // Only supports single, 3D, gpuArray inputs!
    const mxGPUArray *img = mxGPUCreateFromMxArray(prhs[0]);
    const mxGPUArray *ker = mxGPUCreateFromMxArray(prhs[1]);
    if (mxGPUGetClassID(img) != mxSINGLE_CLASS || mxGPUGetNumberOfDimensions(img) != 3)
        mexErrMsgIdAndTxt("conv3d_mex:Input", "Input must be 3D gpuArray single.");
    if (mxGPUGetClassID(ker) != mxSINGLE_CLASS || mxGPUGetNumberOfDimensions(ker) != 3)
        mexErrMsgIdAndTxt("conv3d_mex:Kernel", "Kernel must be 3D gpuArray single.");

    const mwSize *isz = mxGPUGetDimensions(img);
    const mwSize *ksz = mxGPUGetDimensions(ker);

    int nx = (int)isz[0], ny = (int)isz[1], nz = (int)isz[2];
    int kx = (int)ksz[0], ky = (int)ksz[1], kz = (int)ksz[2];

    const float *d_img = (const float*)mxGPUGetDataReadOnly(img);
    const float *d_ker = (const float*)mxGPUGetDataReadOnly(ker);

    mxGPUArray *out = mxGPUCreateGPUArray(3, isz, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    float *d_out = (float*)mxGPUGetData(out);

    dim3 block(8, 8, 4);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y, (nz+block.z-1)/block.z);

    conv3d_single<<<grid, block>>>(d_img, d_ker, d_out, nx, ny, nz, kx, ky, kz);
    hipDeviceSynchronize();

    plhs[0] = mxGPUCreateMxArrayOnGPU(out);

    mxGPUDestroyGPUArray(img);
    mxGPUDestroyGPUArray(ker);
    mxGPUDestroyGPUArray(out);
}
